#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <stdio.h>

#include "omislib.cuh"

void dotProduct(float* c, struct vec* a, struct vec* b)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipblasHandle_t handle;

    hipSetDevice(0);

    hipblasCreate(&handle);
    
    hipMalloc((void**)&dev_a, a->size * sizeof(float));
    hipMalloc((void**)&dev_b, b->size * sizeof(float));
    hipMalloc((void**)&dev_c, sizeof(float));

    hipMemcpy(dev_a, a->val, a->size *  sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b->val, b->size * sizeof(float), hipMemcpyHostToDevice);

    hipblasStatus_t stat = hipblasSdot(handle, a->size, dev_a, 1, dev_b, 1, dev_c);

    hipMemcpy(c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
     
    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);
}