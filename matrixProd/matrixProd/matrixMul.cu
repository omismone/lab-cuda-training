#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include<stdio.h>

#include "omislib.cuh"

__global__ void mulKernel(int* c, int* a, int* b, unsigned int  size)
{
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.y;

    int tmp = 0;
    for (int k = 0; k < size; k++) {
        tmp += a[i * size + k] * b[k * size + j];
    }
    c[i * size + j] = tmp;
}

// Helper function for using CUDA to add arrtors in parallel.
void matrixMul(struct mat* c, struct mat* a, struct mat* b)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    // Allocate GPU buffers for three arrtors (two input, one output)    .
    hipMalloc((void**)&dev_c, c->size[0] * c->size[1] * sizeof(int));

    hipMalloc((void**)&dev_a, a->size[0] * a->size[1] * sizeof(int));

    hipMalloc((void**)&dev_b, b->size[0] * b->size[1] * sizeof(int));

    // Copy input arrtors from host memory to GPU buffers.
    hipMemcpy(dev_a, a->val, a->size[0] * a->size[1] * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_b, b->val, b->size[0] * b->size[1] * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(c->size[0], c->size[1], 1);
    dim3 dimGrid(1, 1, 1);                                                     //to do: roba con pi� blocchi(tile). stesso concetto dello scorso es ma in 2 dimensioni.

    // Launch a kernel on the GPU with one thread for each element.
    mulKernel << <dimGrid, dimBlock >> > (dev_c, dev_a, dev_b, c->size[0]);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output arrtor from GPU buffer to host memory.
    hipMemcpy(c->val, dev_c, c->size[0] * c->size[1] * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}