#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <math.h>

#include "omislib.cuh"



__global__ void addKernel(int* c, int* a, int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add arrtors in parallel.
int addWithCuda(struct arr* c, struct arr* a, struct arr* b)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    // Allocate GPU buffers for three arrtors (two input, one output)    .
    hipMalloc((void**)&dev_c, c->size * sizeof(int));

    hipMalloc((void**)&dev_a, a->size * sizeof(int));

    hipMalloc((void**)&dev_b, b->size * sizeof(int));

    // Copy input arrtors from host memory to GPU buffers.
    hipMemcpy(dev_a, a->val, a->size * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_b, b->val, b->size * sizeof(int), hipMemcpyHostToDevice);

    const unsigned int THREADPERBLOCK = 5;                                              // what if this value exceed max thread per block value?????????????
    dim3 dimBlock(THREADPERBLOCK, 1, 1);
    dim3 dimGrid(ceil(c->size / THREADPERBLOCK), 1, 1);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<dimGrid, dimBlock >> > (dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output arrtor from GPU buffer to host memory.
    hipMemcpy(c->val, dev_c, c->size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}