#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

#include "omislib.cuh"



void productMatVec(struct vec* c, struct mat* a, struct vec* b)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipblasHandle_t handle;

    hipSetDevice(0);

    hipblasCreate(&handle);

    hipMalloc((void**)&dev_a, a->size[0] * a->size[1] * sizeof(float));
    hipMalloc((void**)&dev_b, b->size * sizeof(float));
    hipMalloc((void**)&dev_c, c->size * sizeof(float));

    hipMemcpy(dev_a, a->val, a->size[0] * a->size[1] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b->val, b->size * sizeof(float), hipMemcpyHostToDevice);
    
    const float alpha = 1.0;
    const float beta = 0.0;
    hipblasStatus_t stat = hipblasSgemv(handle, HIPBLAS_OP_T, a->size[1], a->size[0] , &alpha, dev_a, a->size[1], dev_b, 1, &beta, dev_c, 1);

    hipMemcpy(c->val, dev_c, c->size * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

void productMatMat(struct mat* c, struct mat* a, struct mat* b)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipblasHandle_t handle;
    
    hipSetDevice(0);

    hipblasCreate(&handle);

    hipMalloc((void**)&dev_a, a->size[0] * a->size[1] * sizeof(float));
    hipMalloc((void**)&dev_b, b->size[0] * b->size[1] * sizeof(float));
    hipMalloc((void**)&dev_c, c->size[0] * c->size[1] * sizeof(float));

    hipMemcpy(dev_a, a->val, a->size[0] * a->size[1] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b->val, b->size[0] * b->size[1] * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0;
    const float beta = 0.0;
    int m = b->size[1];
    int n = a->size[0];
    int k = b->size[0];
    hipblasStatus_t stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n, dev_a, k, &beta, dev_c, n);

    hipMemcpy(c->val, dev_c, c->size[0] * c->size[1] * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);    
    hipFree(dev_c);
}